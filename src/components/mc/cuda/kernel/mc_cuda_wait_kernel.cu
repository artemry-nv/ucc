#include "hip/hip_runtime.h"
/**
 * Copyright (C) Mellanox Technologies Ltd. 2021.  ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include "../mc_cuda.h"
#ifdef __cplusplus
}
#endif

__global__ void wait_kernel(volatile uint32_t *status) {
    ucc_status_t st;
    *status = UCC_MC_CUDA_TASK_STARTED;
    do {
        st = (ucc_status_t)*status;
    } while(st != UCC_MC_CUDA_TASK_COMPLETED);
    return;
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t ucc_mc_cuda_post_kernel_stream_task(uint32_t *status,
                                                 hipStream_t stream)
{
    wait_kernel<<<1, 1, 0, stream>>>(status);
    CUDACHECK(hipGetLastError());
    return UCC_OK;
}

#ifdef __cplusplus
}
#endif
